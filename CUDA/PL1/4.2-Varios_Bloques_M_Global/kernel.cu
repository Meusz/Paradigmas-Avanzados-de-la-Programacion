#include "hip/hip_runtime.h"
 
#include "stdio.h"  
#include "stdlib.h"



#define FILTER_WIDTH 3
#define WIDTH 16
#define CONVULTION_WIDTH WIDTH+2
#define TILE_WIDTH 8

void mostrar_matriz_inicial(int matriz[WIDTH][WIDTH]);
void mostrar_matriz_filtro(int matriz[FILTER_WIDTH][FILTER_WIDTH]);


__global__ void GPU_mediante_un_bloque_y_memoria_global(int *Matriz, int *filtro, int *Salida) {
	//Se definen los ejes x e y de la matriz de entrada

	int y = blockIdx.y * TILE_WIDTH + threadIdx.y;
	int x = blockIdx.x * TILE_WIDTH + threadIdx.x;

	int filtrox = 1;
	int filtroy = 1;

	int Pvalue = 0;
	

	//Necesitamos acceder a una posicion dentro del area permitida
		for (int i = -1; i <= 1; i++)
		{
			if ((y + i) >= WIDTH || (y + i) < 0) {
				Pvalue += 0;
			}
			else {
				for (int j = -1; j <= 1; j++)
				{
					if ((x + j) >= WIDTH || (x + j) < 0) {
						Pvalue += 0;
					}
					else {
						Pvalue += Matriz[ (y + i) * WIDTH + (x + j)] *  filtro[(filtroy + i) * FILTER_WIDTH + (filtrox + j)];
					}
				}
			}
		}
	
	Salida[y * WIDTH + x] = Pvalue;
}



int main() {



	int A[WIDTH][WIDTH] = { 0 };
	int Salida[WIDTH][WIDTH] = { 0 };
	int filtro[FILTER_WIDTH][FILTER_WIDTH] = { 0 };
	int invertida[FILTER_WIDTH][FILTER_WIDTH] = { 0 };
	//int invertida[FILTER_WIDTH][FILTER_WIDTH] = { {1,2,3},{4,5,6},{7,8,9} };
	//Declaramos los bloques dim
	dim3 DimGrid(WIDTH / TILE_WIDTH, WIDTH / TILE_WIDTH);
	dim3 DimBlock(TILE_WIDTH, TILE_WIDTH);

	//Reservamos la memoria necesaria
	int* d_A;
	int* d_salida;
	int* d_filtro;

	size_t size = WIDTH * WIDTH * sizeof(int);
	size_t size_filtro = FILTER_WIDTH * FILTER_WIDTH * sizeof(int);

	hipMalloc(&d_A, size);

	hipMalloc(&d_salida, size);

	hipMalloc(&d_filtro, size);

	//-------------------------------------------------------- -
		//Rellenamos la matriz A de numeros aleatorios
		//1 + rand() % (99)

		for (int i = 0; i < WIDTH; i++) {
			for (int j = 0; j < WIDTH; j++) {
				A[i][j] = 1 + rand() % (99);
			}
		}
	//Mostramos la matriz inicial
		printf("La matriz A queda como \n: ");

	mostrar_matriz_inicial(A);

	//-------------------------------------------------------- -
		//Rellenamos la matriz Filtro de numeros aleatorios de 0 a 1


		for (int i = 0; i < FILTER_WIDTH; i++) {
			for (int j = 0; j < FILTER_WIDTH; j++) {
				filtro[i][j] = rand() % (2);
			}
		}
	//Mostramos la matriz filtro
		printf("La matriz es filtro queda como \n" );

	mostrar_matriz_filtro(filtro);

	//Calculamos la invertida de la matriz filtro
		int a = 0;
	for (int i = FILTER_WIDTH - 1; i >= 0; i--) {
		int b = 0;
		for (int j = FILTER_WIDTH - 1; j >= 0; j--) {
			invertida[i][j] = filtro[a][b];
			b++;
		}
		a++;
	}
	//Mostramos la matriz filtro invertida
	printf("La invertida de la matriz filtro queda como \n");
	mostrar_matriz_filtro(invertida);


	//Setup the execution configuration +
	//	Cargamos las matrices en la GPU



	hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
	hipMemcpy(d_salida, Salida, size, hipMemcpyHostToDevice);
	hipMemcpy(d_filtro, invertida, size_filtro, hipMemcpyHostToDevice);


	//Declaramos que habra un solo grid



	GPU_mediante_un_bloque_y_memoria_global  <<< DimGrid, DimBlock >>> (d_A, d_filtro, d_salida);

	//Realizadas las operaciones copiamos los resultados en CPU

	hipMemcpy(Salida, d_salida, size, hipMemcpyDeviceToHost);

	hipFree(d_A);
	hipFree(d_salida);
	hipFree(d_filtro);


	printf("La matriz convulcionada queda como \n");

	mostrar_matriz_inicial(Salida);

	getchar();
}


void mostrar_matriz_inicial(int matriz[WIDTH][WIDTH]) {
	for (int i = 0; i < WIDTH; i++) {
		for (int j = 0; j < WIDTH; j++) {
			printf("%d \t", matriz[i][j]);
		}
		printf("\n");
	}
	printf("\n");
}

void mostrar_matriz_filtro(int matriz[FILTER_WIDTH][FILTER_WIDTH]) {
	for (int i = 0; i < FILTER_WIDTH; i++) {
		for (int j = 0; j < FILTER_WIDTH; j++) {
			printf("%d \t", matriz[i][j]);
		}
		printf("\n");
	}
	printf("\n");
}
