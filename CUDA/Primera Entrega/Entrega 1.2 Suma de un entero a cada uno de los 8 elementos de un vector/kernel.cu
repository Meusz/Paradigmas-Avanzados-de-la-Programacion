#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdio.h> #include <stdlib .h>
#include <hip/hip_runtime.h> #include <cuda runtime.h>

__global__ void add2(int ∗a) {
    int i = threadIdx.x;
    a[i] = a[i] + 8;
}

int main() {
    const int N = 8;
    int a[N] = {0,2,43,21,22,45,12,23};
    size_t size = N * sizeof(int);
    int* a_d;
    
    hipMalloc(&a_d, size);
    hipMemcpy(a_d, a, size, hipMemcpyHostToDevice);
    add2<< <1, 8 >> > (a_d);
    hipMemcpy(a, a_d, size, hipMemcpyDeviceToHost);
    for (int i = 0; i < N; i++) {
        printf("a = %d\n", a[i]);
    }
    hipFree(a_d);
}