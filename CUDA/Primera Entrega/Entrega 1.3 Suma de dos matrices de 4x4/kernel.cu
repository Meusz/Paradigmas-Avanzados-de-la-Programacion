﻿#include "hip/hip_runtime.h"
 
#include <stdio.h> 
#include <stdio.h>  
#include <stdlib.h>
 

__global__ void suma_matrices(int *matriz1, int *matriz2, int *final) {
	
	int i= blockIdx.x + threadIdx.x * blockDim.x;

	final[i] = matriz1[i] + matriz2[i];
}


int main() {
	const int N = 4;
	int matriz1[N][N] = { {132,213,22,331},{372,7245,72,722},{2574,222,775,75},{1,25,2,4} };
	int matriz2[N][N] = { {457,225,244,222},{976,257,7456,6467},{5473,543,566,456},{8,365,356,6} };
	int final[N][N] = { 0 };
	int* g_final;
	int* g_matriz1;
	int* g_matriz2;

	size_t size = N * N * sizeof(int);

	hipMalloc(&g_final, size);
	hipMalloc(&g_matriz1, size);
	hipMalloc(&g_matriz2, size);

	hipMemcpy(g_matriz1, matriz1, size, hipMemcpyHostToDevice);
	hipMemcpy(g_matriz2, matriz2, size, hipMemcpyHostToDevice);
	hipMemcpy(g_final, final, size, hipMemcpyHostToDevice);

	suma_matrices <<<4,4>>> (g_matriz1, g_matriz2, g_final);

	hipMemcpy(final, g_final, size, hipMemcpyDeviceToHost);

	hipFree(g_matriz1);
	hipFree(g_matriz2);
	hipFree(g_final);

	//mostramos el resultado por pantalla

	printf("El resultado de la suma es: \n");

	for (int i = 0; i < N; i++) {
		for (int j = 0; j < N; j++) {
			printf("%d \t", final[i][j]);
		}
		printf("\n");
	}
	printf("\n");

}
