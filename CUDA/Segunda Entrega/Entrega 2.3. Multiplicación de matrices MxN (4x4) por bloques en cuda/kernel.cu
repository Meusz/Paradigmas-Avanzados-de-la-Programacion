#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "" 
#include <stdio.h>  
#include <stdlib.h>
#include <hip/hip_runtime.h> 

__global__ void multi_matrices(int *a, int *b, int *c, int Width) {

	int Pvalue = 0;

	int y = blockIdx.y * Width + threadIdx.y;

	int x = blockIdx.x * Width + threadIdx.x;

	for (int k = 0; k < Width; ++k) {

		int Melement = a[y * Width + k];

		int Nelement = b[k*Width + x];

		Pvalue += Melement * Nelement;
	}

	c[y*Width + x] = Pvalue;
}


int main() {
	const int total_witdh = 16;
	const int Width = 4;
	int a[Width][Width] = { {2,3,4,7},{8,4,1,4},{6,7,4,9},{4,6,1,1} };
	int b[Width][Width] = { {3,4,5,1},{7,8,9,10},{1,3,6,8},{9,8,5,3} };
	int c[Width][Width] = { 0 };
	int* d_c;
	int* d_b;
	int* d_a;

	size_t size = Width * Width * sizeof(int);

	hipMalloc(&d_a, size);
	hipMalloc(&d_b, size);
	hipMalloc(&d_c, size);

	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
	hipMemcpy(d_c, c, size, hipMemcpyHostToDevice);

	// Setup the execution configuration+
	//Declaramos que habra un solo grid
	dim3 dimGrid(1, 1);
	//Declaramos que habra 4 bloques de 4 hilos
	dim3 dimBlock(4, 4);

	multi_matrices << < dimGrid, dimBlock >> > (d_a, d_b, d_c, Width);

	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

	hipFree(d_c);
	hipFree(d_b);
	hipFree(d_a);

	//mostramos el resultado por pantalla

	printf("El resultado de la suma es: \n");

	for (int i = 0; i < Width; i++) {
		for (int j = 0; j < Width; j++) {
			printf("%d \t", c[i][j]);
		}
		printf("\n");
	}
	printf("\n");

}
