#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "" 
#include <stdio.h>  
#include <stdlib.h>
#include <hip/hip_runtime.h> 

__global__ void multi_matrices(int *a, int *c, int Width) {

	int Pvalue = 0;

	int y = blockIdx.y * Width + threadIdx.y;

	int x = blockIdx.x * Width + threadIdx.x;

	

	c[y*Width + x] = a[x*Width + y];
}


int main() {
	const int Width = 4;
	const int a[Width][Width] = { {2,3,4,7},{8,4,1,4},{6,7,4,9},{4,6,1,1} };
	int c[Width][Width] = { 0 };
	int* d_c;
	int* d_a;

	size_t size = Width * Width * sizeof(int);

	hipMalloc(&d_a, size);
	hipMalloc(&d_c, size);

	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_c, c, size, hipMemcpyHostToDevice);

	// Setup the execution configuration+
	//Declaramos que habra un solo grid
	dim3 dimGrid(1, 1);
	//Declaramos que habra 4 bloques de 4 hilos
	dim3 dimBlock(4, 4);

	multi_matrices << < dimGrid, dimBlock >> > (d_a, d_c, Width);

	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

	hipFree(d_c);
	hipFree(d_a);

	//mostramos el resultado por pantalla
	printf("La matriz inicial es: \n");

	for (int i = 0; i < Width; i++) {
		for (int j = 0; j < Width; j++) {
			printf("%d \t", a[i][j]);
		}
		printf("\n");
	}
	printf("\n");

	printf("El resultado de la traspuesta es: \n");

	for (int i = 0; i < Width; i++) {
		for (int j = 0; j < Width; j++) {
			printf("%d \t", c[i][j]);
		}
		printf("\n");
	}
	printf("\n");

}