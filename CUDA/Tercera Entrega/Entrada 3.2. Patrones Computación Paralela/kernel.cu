#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "" 
#include <stdio.h>  
#include <stdlib.h>
#include <hip/hip_runtime.h> 

#define WIDTH 16
#define TILE_WIDTH 8

__device__ int minimo_multiplo(int num) {
	int i = 2;
	while (num % i != 0) {
		i++;
	}
	return i;
}

__global__ void Stencil(int *a, int *c) {
	//Se suman todos los numeros adyacentes al de la casilla central. Los adyacentes se ponen a 0

	int y = blockIdx.y * TILE_WIDTH + threadIdx.y;

	int x = blockIdx.x * TILE_WIDTH + threadIdx.x;

	if (a[y*WIDTH + x] == 9) {
		int Pvalue = a[y*WIDTH + x];
		
		c[(y - 1)*WIDTH + x] = 0;
		Pvalue += a[(y - 1)*WIDTH + x];
		c[(y + 1)*WIDTH + x] = 0;
		Pvalue += a[(y + 1)*WIDTH + x];
		c[y*WIDTH +(x - 1)] = 0;
		Pvalue += a[y*WIDTH + (x - 1)];
		c[y*WIDTH + (x + 1)] = 0;
		Pvalue += a[y*WIDTH + (x + 1)];

		c[y*WIDTH + x] = Pvalue;
	}
	else {
		c[y*WIDTH + x] = a[y*WIDTH + x];
	}
	
}
__global__ void Scatter(int *a, int *c) {
	//Si el numero es 11, suma 11 a todos los numeros en su columna
	int y = blockIdx.y * TILE_WIDTH + threadIdx.y;

	int x = blockIdx.x * TILE_WIDTH + threadIdx.x;

	if (a[y*WIDTH + x] == 11) {

		for (int i = 0; (y - i) >= 0 ; i++) {
			c[(y - i)*WIDTH + x ] = a[(y - i)*WIDTH + x] + 11;
		}
		for (int i = 0; (y + i) < WIDTH; i++)
		{
			c[(y + i)*WIDTH + x ] = a[(y + i)*WIDTH + x ] + 11;
		}

		c[y*WIDTH + x] = 11;
		
				
	}
	else 
	{
		c[y*WIDTH + x] = a[y*WIDTH + x];
	}

}
__global__ void Gather(int *a, int *c) {
	//Si hay una fila con 3 o mas numeros con el minimo comun multiplo, la posicion que mas en el medio esta acumula la suma de los
	//numeros y los demas asumen valor 0

	int y = blockIdx.y * TILE_WIDTH + threadIdx.y;
	int x = blockIdx.x * TILE_WIDTH + threadIdx.x;

	int comun = minimo_multiplo(a[y*WIDTH + x]);
	int izq = 0, der = 0;
	while (comun == minimo_multiplo(a[y*WIDTH + x + der]) ) {
		der++;
	}
	while (x - izq>=0 && comun == minimo_multiplo(a[y*WIDTH + x - izq]) ) {
		izq++;
	}
	izq--; der--;
	if ( (izq - der) ==1 && izq + der>1 ) {
		int Pvalue = 0;
		for (int i = x - izq; i <= x + der; i++)
		{
			Pvalue += a[y*WIDTH + i];
			c[y*WIDTH + i] = 0;
		}
		c[y*WIDTH + x] = Pvalue;
	}
	else {
		c[y*WIDTH + x] = a[y*WIDTH + x];
	}
}

/*__global__ void Gatherf(int *a, int *c) {
	//Si hay una fila con 3 numeros con un minimo comun multiplo, la posicion que mas en el medio esta acumula la suma de los
	//numeros y los demas asumen valor 0
	
	int y = blockIdx.y * TILE_WIDTH + threadIdx.y;
	int x = blockIdx.x * TILE_WIDTH + threadIdx.x;

	int comun = minimo_multiplo(a[y*WIDTH + x]);
	int izq = 0, der = 0;
	while ( x + der < WIDTH && comun == minimo_multiplo(a[y*WIDTH + x + der])) {
		der++;
	}
	while (x - izq >= 0 && comun == minimo_multiplo(a[y*WIDTH + x - izq])) {
		izq++;
	}
	izq--; der--;
	if ( izq + der > 2) {
		if (( (izq - der) == 1 || (izq - der) == 0)) {
			int Pvalue = 0;
			for (int i = 1; i <= der; i++)
			{
				Pvalue += a[y*WIDTH + x+der];
			}
			for (int i = 1; i <= izq; i++)
			{
				Pvalue += a[y*WIDTH + x -izq];
			}
			c[y*WIDTH + x] = Pvalue + a[y*WIDTH + x];
			
			
		}
		else {
		c[y*WIDTH + x] = 0;
		}
	}
	else {
		c[y*WIDTH + x] += a[y*WIDTH + x];
	}
}
__global__ void Gatherc(int *a, int *c) {
	//Si hay una columna con 3 o mas numeros con el minimo comun multiplo, la posicion que mas en el medio esta acumula la suma de los
	//numeros y los demas asumen valor 0
	int Pvalue = 0;
	int y = blockIdx.y * TILE_WIDTH + threadIdx.y;
	int x = blockIdx.x * TILE_WIDTH + threadIdx.x;

	int comun = minimo_multiplo(a[y*WIDTH + x]);
	int abj = 0, ari = 0;
	while (y + ari < WIDTH && comun == minimo_multiplo(a[(y+ ari+1 )*WIDTH + x])) {
		ari++;
	}
	while (y - abj  >= 0 && comun == minimo_multiplo(a[(y- abj-1 )*WIDTH + x])) {
		abj++;
	}
	if (abj + ari > 2) {
		if (((abj - ari) == 1 || (abj - ari) == 0)) {
			c[y*WIDTH + x] = c[y*WIDTH + x] + a[y*WIDTH + x];


		}
		else {
			int diff = ari - abj;
			if ((ari + abj) % 2 != 0 && diff % 2 != 0) {
				diff += 1;

			}
			diff = diff / 2;

			c[y*WIDTH + x] = 0;

			c[(y + diff)*WIDTH + x] = c[(y + diff)*WIDTH + x] + a[y*WIDTH + x];



		}

	}
	else {
		c[y*WIDTH + x] = a[y*WIDTH + x];
	}
}*/


int main() {
	
	
	int a[WIDTH][WIDTH] = { 0 };
	int c[WIDTH][WIDTH] = { 0 };
	int* d_c;
	int* d_b;
	int* d_a;
	int* d_d;
	dim3 DimGrid(WIDTH / TILE_WIDTH, WIDTH / TILE_WIDTH);
	dim3 DimBlock(TILE_WIDTH, TILE_WIDTH);

	size_t size = WIDTH * WIDTH * sizeof(int);

	hipMalloc(&d_a, size);
	hipMalloc(&d_c, size);
	
	/*---------------------------------------------------------
	-----------------------------------------------------------
	-----------------------------------------------------------*/

	//Rellenamos las matrices a y b de numeros aleatorios 
	//1 + rand() % (99)
	for (int i = 0; i < WIDTH; i++) {
		for (int j = 0; j < WIDTH; j++) {
			a[i][j] = 1 + rand() % (99);
		}
	}
	//mostramos el resultado por pantalla
	printf("La matriz inicial es: \n");

	for (int i = 0; i < WIDTH; i++) {
		for (int j = 0; j < WIDTH; j++) {
			printf("%d \t", a[i][j]);
		}
		printf("\n");
	}
	printf("\n");

	/*---------------------------------------------------------
	-----------------------------------------------------------
	-----------------------------------------------------------*/


	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_c, c, size, hipMemcpyHostToDevice);

	// Setup the execution configuration+
	//Declaramos que habra un solo grid
	
	

	Stencil << < DimGrid, DimBlock >> > (d_a, d_c);

	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
	hipFree(d_a);
	hipFree(d_c);
	/*---------------------------------------------------------
	-----------------------------------------------------------
	-----------------------------------------------------------*/
	printf("El resultado de stencil es: \n");
	for (int i = 0; i < WIDTH; i++) {
		for (int j = 0; j < WIDTH; j++) {
			printf("%d \t", c[i][j]);
		}
		printf("\n");
	}
	printf("\n");
	/*---------------------------------------------------------
	-----------------------------------------------------------
	-----------------------------------------------------------*/

	int b[WIDTH][WIDTH] = { 0 };

	hipMalloc(&d_a, size);
	hipMalloc(&d_b, size);

	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
	
	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);


	Scatter << < DimGrid, DimBlock >> > (d_a, d_b);

	hipMemcpy(b, d_b, size, hipMemcpyDeviceToHost);

	/*---------------------------------------------------------
	-----------------------------------------------------------
	-----------------------------------------------------------*/
	printf("El resultado de scatter es: \n");
	for (int i = 0; i < WIDTH; i++) {
		for (int j = 0; j < WIDTH; j++) {
			printf("%d \t", b[i][j]);
		}
		printf("\n");
	}
	printf("\n");
	/*---------------------------------------------------------
	-----------------------------------------------------------
	-----------------------------------------------------------*/
	
	hipFree(d_a);
	hipFree(d_b);

	/*--------------------------------------------------------------*/
	int d[WIDTH][WIDTH] = { 0 };

	hipMalloc(&d_a, size);
	hipMalloc(&d_d, size);
	
	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_d, d, size, hipMemcpyHostToDevice);


	Gather << < DimGrid, DimBlock >> > (d_a, d_b);

	hipMemcpy(d, d_d, size, hipMemcpyDeviceToHost);

	/*---------------------------------------------------------
	-----------------------------------------------------------
	-----------------------------------------------------------*/
	printf("El resultado de Gatherc es: \n");
	for (int i = 0; i < WIDTH; i++) {
		for (int j = 0; j < WIDTH; j++) {
			printf("%d \t", d[i][j]);
		}
		printf("\n");
	}
	printf("\n");
	/*---------------------------------------------------------
	-----------------------------------------------------------
	-----------------------------------------------------------*/

	hipFree(d_a);
	hipFree(d_d);
}