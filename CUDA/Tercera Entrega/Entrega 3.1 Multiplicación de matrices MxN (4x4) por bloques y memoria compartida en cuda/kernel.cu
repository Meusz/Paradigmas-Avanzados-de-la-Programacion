#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "" 
#include <stdio.h>  
#include <stdlib.h>
#include <hip/hip_runtime.h> 


const int total_witdh = 16;
const int Width = 4;
const int TILE_WIDTH = 2;

__global__ void multi_matrices(int *a, int *b, int *c, int Width) {
	
	__shared__ int Mds[TILE_WIDTH][TILE_WIDTH];
	__shared__ int Nds[TILE_WIDTH][TILE_WIDTH];

	int bx = blockIdx.x; int by = blockIdx.y;
	int tx = threadIdx.x; int ty = threadIdx.y;


	// Identify the row and column of
	// the Pd element to work on

	int Row = by * TILE_WIDTH + ty;
	int Col = bx * TILE_WIDTH + tx;
	int Pvalue = 0;
	// Loop over the Md and Nd tiles required
	// to compute the Pd element
	for (int m = 0; m < Width / TILE_WIDTH; ++m) {
		// Collaborative loading of Md and Nd
		// tiles into shared memory
		Mds[ty][tx] = a[Row*Width + (m*TILE_WIDTH + tx)];
		Nds[ty][tx] = b[(m*TILE_WIDTH + ty)*Width + Col];

		__syncthreads();
		for (int k = 0; k < TILE_WIDTH; ++k)
			Pvalue += Mds[ty][k] * Nds[k][tx];
		__syncthreads();
	}
	c[Row*Width + Col] = Pvalue;

}


int main() {
	
	int a[Width][Width] = { {2,3,4,7},{8,4,1,4},{6,7,4,9},{4,6,1,1} };
	int b[Width][Width] = { {3,4,5,1},{7,8,9,10},{1,3,6,8},{9,8,5,3} };
	int c[Width][Width] = { 0 };
	int* d_c;
	int* d_b;
	int* d_a;

	size_t size = Width * Width * sizeof(int);

	hipMalloc(&d_a, size);
	hipMalloc(&d_b, size);
	hipMalloc(&d_c, size);

	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
	hipMemcpy(d_c, c, size, hipMemcpyHostToDevice);

	// Setup the execution configuration+
	//Declaramos que habra un solo grid
	dim3 dimBlock(TILE_WIDTH, TILE_WIDTH);

	dim3 dimGrid(Width / TILE_WIDTH, Width / TILE_WIDTH);


	multi_matrices << < dimGrid, dimBlock >> > (d_a, d_b, d_c, Width);

	//MATRIZ ORIGINAL
	printf("La matriz original es: \n");

	for (int i = 0; i < Width; i++) {
		for (int j = 0; j < Width; j++) {
			printf("%d \t", c[i][j]);
		}
		printf("\n");
	}
	printf("\n\n");

	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

	hipFree(d_c);
	hipFree(d_b);
	hipFree(d_a);

	//mostramos el resultado por pantalla

	printf("El resultado de la multiplicacion es: \n");

	for (int i = 0; i < Width; i++) {
		for (int j = 0; j < Width; j++) {
			printf("%d \t", c[i][j]);
		}
		printf("\n");
	}
	printf("\n");

}